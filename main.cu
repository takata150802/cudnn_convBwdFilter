#include <iostream>
#include <vector>
#include <random>
#include <limits>
#include <cassert>
#include <hipDNN.h>

#define CHECK(call)                                                  \
{                                                                    \
    const hipError_t error = call;                                  \
    std::cout << "CHECK hipError_t: ";                              \
    if (error != hipSuccess)                                        \
    {                                                                \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "Error"                                         \
                  << std::endl;                                      \
        std::cout << "code: "                                        \
                  << error                                           \
                  << ", "                                            \
                  << "reason: "                                      \
                  << hipGetErrorString(error)                       \
                  << std::endl;                                      \
        std::exit(EXIT_FAILURE);                                     \
    }                                                                \
    else                                                             \
    {                                                                \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "hipSuccess"                                   \
                  << std::endl;                                      \
    }                                                                \
}

#define checkCUDNN(call)                                             \
{                                                                    \
    hipdnnStatus_t status = (call);                                   \
    if (status != HIPDNN_STATUS_SUCCESS) {                            \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "Error"                                         \
                  << std::endl;                                      \
        std::cout << "code: "                                        \
                  << status                                          \
                  << ", "                                            \
                  << "reason: "                                      \
                  << hipdnnGetErrorString(status)                     \
                  << std::endl;                                      \
        std::exit(EXIT_FAILURE);                                     \
    }                                                                \
}

void rand_vector_float (std::vector<float> &v);
namespace {
    const char* getAlgoName(hipdnnConvolutionBwdFilterAlgo_t algo);
}

int main(int argc, char *argv[]) {
    hipdnnHandle_t handle;
    checkCUDNN(hipdnnCreate(&handle));
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    const int n = 5, ci = 4, hi = 1, wi = 1, 
          pad_h = 1, pad_w = 1, u = 1, v = 1, 
          dilation_h = 1, dilation_w = 1,
          co = 3, kernel_h = 1, kernel_w = 1;
    int n_dmy, co_dmy, ho, wo;

    hipdnnConvolutionDescriptor_t convDesc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(
                /*hipdnnConvolutionDescriptor_t*/ convDesc,
                /*int                         */ pad_h,
                /*int                         */ pad_w,
                /*int                         */ u,
                /*int                         */ v,
                /*int                         */ dilation_h,
                /*int                         */ dilation_w,
                /*hipdnnConvolutionMode_t      */ HIPDNN_CROSS_CORRELATION,
                /*hipdnnDataType_t             */ HIPDNN_DATA_FLOAT));

    hipdnnTensorDescriptor_t xDesc;
    hipdnnCreateTensorDescriptor(&xDesc);
    checkCUDNN(hipdnnSetTensor4dDescriptor(
                /*hipdnnTensorDescriptor_t*/ xDesc,
                /*hipdnnTensorFormat_t    */ HIPDNN_TENSOR_NCHW,
                /*hipdnnDataType_t        */ HIPDNN_DATA_FLOAT,
                /*int                    */ n,
                /*int                    */ ci,
                /*int                    */ hi,
                /*int                    */ wi));


    hipdnnFilterDescriptor_t dwDesc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&dwDesc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(
                /*hipdnnFilterDescriptor_t*/ dwDesc,
                /*hipdnnDataType_t        */ HIPDNN_DATA_FLOAT,
                /*hipdnnTensorFormat_t    */ HIPDNN_TENSOR_NCHW,
                /*int                    */ co,
                /*int                    */ ci,
                /*int                    */ kernel_h,
                /*int                    */ kernel_w));

    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(
                /*const hipdnnConvolutionDescriptor_t*/ convDesc,
                /*const hipdnnTensorDescriptor_t     */ xDesc,
                /*const hipdnnFilterDescriptor_t     */ dwDesc,
                /*int*                              */ &n_dmy,
                /*int*                              */ &co_dmy,
                /*int*                              */ &ho,
                /*int*                              */ &wo));
    assert(n == n_dmy);
    assert(co == co_dmy);

    hipdnnTensorDescriptor_t dyDesc;
    hipdnnCreateTensorDescriptor(&dyDesc);
    checkCUDNN(hipdnnSetTensor4dDescriptor(
                /*hipdnnTensorDescriptor_t*/ dyDesc,
                /*hipdnnTensorFormat_t    */ HIPDNN_TENSOR_NCHW,
                /*hipdnnDataType_t        */ HIPDNN_DATA_FLOAT,
                /*int                    */ n,
                /*int                    */ co,
                /*int                    */ ho,
                /*int                    */ wo));

    hipdnnConvolutionBwdFilterAlgo_t algo;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(
                /*hipdnnHandle_t                        */ handle,
                /*const hipdnnTensorDescriptor_t        */ xDesc,
                /*const hipdnnTensorDescriptor_t        */ dyDesc,
                /*const hipdnnConvolutionDescriptor_t   */ convDesc,
                /*const hipdnnFilterDescriptor_t        */ dwDesc,
                /*hipdnnConvolutionBwdFilterPreference_t*/ HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
                /*size_t (is not used)                 */ 0,
                /*hipdnnConvolutionBwdFilterAlgo_t      */ &algo));
    std::cout << "hipdnnConvolutionBwdFilterAlgo_t: " << getAlgoName(algo) << std::endl;

    size_t workSpaceSizeInBytes;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
                /*hipdnnHandle_t                     */ handle,
                /*const hipdnnTensorDescriptor_t     */ xDesc,
                /*const hipdnnTensorDescriptor_t     */ dyDesc,
                /*const hipdnnConvolutionDescriptor_t*/ convDesc,
                /*const hipdnnFilterDescriptor_t     */ dwDesc,
                /*hipdnnConvolutionBwdFilterAlgo_t   */ algo,
                /*size_t                            */ &workSpaceSizeInBytes));

    std::vector<float> h_x(n * ci * hi * wi, 0);
    std::vector<float> h_dy(n * co * ho * wo, 0);
    std::vector<float> h_dw(co * ci * kernel_h * kernel_w,
            std::numeric_limits<float>::quiet_NaN());
    std::vector<float> h_dw_expct(co * ci * kernel_h * kernel_w,
            std::numeric_limits<float>::quiet_NaN());
    rand_vector_float(h_x);
    rand_vector_float(h_dy);

    void *x = nullptr, *dy = nullptr, *dw = nullptr, *workSpace = nullptr;
    size_t size_x = n * ci * hi * wi * sizeof(float);
    size_t size_dy = n * co * ho * wo * sizeof(float);
    size_t size_dw = co * ci * kernel_h * kernel_w * sizeof(float);
    hipMalloc (&x, size_x);
    hipMalloc (&dy, size_dy);
    hipMalloc (&dw, size_dw);
    hipMalloc (&workSpace, workSpaceSizeInBytes);
    hipMemset (dw, 0xff, size_dw);

    hipMemcpy(x, h_x.data(), size_x, hipMemcpyHostToDevice);
    hipMemcpy(dy, h_dy.data(), size_dy, hipMemcpyHostToDevice);

    const float alpha = 1, beta = 0;
    hipEventRecord(start);
    checkCUDNN(hipdnnConvolutionBackwardFilter(
                /*hipdnnHandle_t                     */ handle,
                /*const void *                      */ &alpha,
                /*const hipdnnTensorDescriptor_t     */ xDesc,
                /*const void *                      */ x,
                /*const hipdnnTensorDescriptor_t     */ dyDesc,
                /*const void *                      */ dy,
                /*const hipdnnConvolutionDescriptor_t*/ convDesc,
                /*hipdnnConvolutionBwdFilterAlgo_t   */ algo,
                /*void *                            */ workSpace,
                /*size_t                            */ workSpaceSizeInBytes,
                /*const void *                      */ &beta,
                /*const hipdnnFilterDescriptor_t     */ dwDesc,
                /*void *                            */ dw));
    hipEventRecord(stop);

    CHECK(hipDeviceSynchronize());
    hipMemcpy(h_dw.data(), dw, size_dw, hipMemcpyDeviceToHost);

    for (std::vector<float>::const_iterator i = h_dw.begin(); i != h_dw.end(); ++i)
        std::cout << *i << ' ';
    std::cout << std::endl;

    for (std::vector<float>::const_iterator i = h_dw_expct.begin(); i != h_dw_expct.end(); ++i)
        std::cout << *i << ' ';
    std::cout << std::endl;

    float msec = 0;
    hipEventElapsedTime(&msec, start, stop);
    std::cout << "Exec time: " << msec * 1000 << "[usec]" << std::endl;

    hipFree(x);
    hipFree(dy);
    hipFree(dw);
    hipFree(workSpace);
    hipdnnDestroyTensorDescriptor(xDesc);
    hipdnnDestroyTensorDescriptor(dyDesc);
    hipdnnDestroyFilterDescriptor(dwDesc);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipdnnDestroy(handle);
    CHECK(hipDeviceSynchronize());
    return 0;
}

std::mt19937 mt(0);
void rand_vector_float (std::vector<float> &v) {
    std::normal_distribution<> rand(0, 5);
    for (std::vector<float>::iterator i = v.begin(); i != v.end(); ++i) {
        *i = rand(mt);
    }
    return;
}

namespace {
    const char* getAlgoName(hipdnnConvolutionBwdFilterAlgo_t algo) 
    {
        switch (algo) 
        {
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0"; /* non-deterministic */
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1";
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT";
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_3: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_3"; /* non-deterministic */
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_WINOGRAD: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_WINOGRAD"; /* not implemented */
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_WINOGRAD_NONFUSED: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_WINOGRAD_NONFUSED";
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT_TILING: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT_TILING";
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT";
            default: std::exit(EXIT_FAILURE); return "Error";
        }
    }
}
