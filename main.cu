#include "hip/hip_runtime.h"
/**
 * Example usage of cuDNN convolution backward filter
 * CUDA Version: 10.0
 * cuDNN Version: 7.4
 *
 * @author     ryotakata
 **/
#include <algorithm>
#include <iostream>
#include <vector>
#include <random>
#include <limits>
#include <cassert>
#include <hipDNN.h>

#define CHECK(call)                                                  \
{                                                                    \
    const hipError_t error = call;                                  \
    std::cout << "CHECK hipError_t: ";                              \
    if (error != hipSuccess)                                        \
    {                                                                \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "Error"                                         \
                  << std::endl;                                      \
        std::cout << "code: "                                        \
                  << error                                           \
                  << ", "                                            \
                  << "reason: "                                      \
                  << hipGetErrorString(error)                       \
                  << std::endl;                                      \
        std::exit(EXIT_FAILURE);                                     \
    }                                                                \
    else                                                             \
    {                                                                \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "hipSuccess"                                   \
                  << std::endl;                                      \
    }                                                                \
}

#define checkCUDNN(call)                                             \
{                                                                    \
    hipdnnStatus_t status = (call);                                   \
    if (status != HIPDNN_STATUS_SUCCESS) {                            \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "Error"                                         \
                  << std::endl;                                      \
        std::cout << "code: "                                        \
                  << status                                          \
                  << ", "                                            \
                  << "reason: "                                      \
                  << hipdnnGetErrorString(status)                     \
                  << std::endl;                                      \
        std::exit(EXIT_FAILURE);                                     \
    }                                                                \
}

void rand_vector_float (std::vector<float> &v);
namespace {
    const char* getAlgoName(hipdnnConvolutionBwdFilterAlgo_t algo);
}
void pseudoConvolutionBackwardFilter(
        const std::vector<float> &x, 
        const std::vector<float> &dy,
        std::vector<float> &dw,
        const int N, const int  Ci, const int Hi, const int Wi,
        const int Co, const int Ho, const int Wo,
        const int Hk, const int Wk, const int Hs, const int Ws,
        const int Hp, const int Wp);
float getMaxAbsError(const std::vector<float> &exp, const std::vector<float> &act);

int main(int argc, char *argv[]) {
    hipdnnHandle_t handle;
    checkCUDNN(hipdnnCreate(&handle));
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    const int n = 32, ci = 2, hi = 14, wi = 14, 
          pad_h = 0, pad_w = 0, u = 1, v = 1, 
          dilation_h = 1, dilation_w = 1,
          co = 3, kernel_h = 1, kernel_w = 1;
    int n_dmy, co_dmy, ho, wo;

    hipdnnConvolutionDescriptor_t convDesc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(
                /*hipdnnConvolutionDescriptor_t*/ convDesc,
                /*int                         */ pad_h,
                /*int                         */ pad_w,
                /*int                         */ u,
                /*int                         */ v,
                /*int                         */ dilation_h,
                /*int                         */ dilation_w,
                /*hipdnnConvolutionMode_t      */ HIPDNN_CROSS_CORRELATION,
                /*hipdnnDataType_t             */ HIPDNN_DATA_FLOAT));

    hipdnnTensorDescriptor_t xDesc;
    hipdnnCreateTensorDescriptor(&xDesc);
    checkCUDNN(hipdnnSetTensor4dDescriptor(
                /*hipdnnTensorDescriptor_t*/ xDesc,
                /*hipdnnTensorFormat_t    */ HIPDNN_TENSOR_NCHW,
                /*hipdnnDataType_t        */ HIPDNN_DATA_FLOAT,
                /*int                    */ n,
                /*int                    */ ci,
                /*int                    */ hi,
                /*int                    */ wi));


    hipdnnFilterDescriptor_t dwDesc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&dwDesc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(
                /*hipdnnFilterDescriptor_t*/ dwDesc,
                /*hipdnnDataType_t        */ HIPDNN_DATA_FLOAT,
                /*hipdnnTensorFormat_t    */ HIPDNN_TENSOR_NCHW,
                /*int                    */ co,
                /*int                    */ ci,
                /*int                    */ kernel_h,
                /*int                    */ kernel_w));

    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(
                /*const hipdnnConvolutionDescriptor_t*/ convDesc,
                /*const hipdnnTensorDescriptor_t     */ xDesc,
                /*const hipdnnFilterDescriptor_t     */ dwDesc,
                /*int*                              */ &n_dmy,
                /*int*                              */ &co_dmy,
                /*int*                              */ &ho,
                /*int*                              */ &wo));
    assert(n == n_dmy);
    assert(co == co_dmy);

    hipdnnTensorDescriptor_t dyDesc;
    hipdnnCreateTensorDescriptor(&dyDesc);
    checkCUDNN(hipdnnSetTensor4dDescriptor(
                /*hipdnnTensorDescriptor_t*/ dyDesc,
                /*hipdnnTensorFormat_t    */ HIPDNN_TENSOR_NCHW,
                /*hipdnnDataType_t        */ HIPDNN_DATA_FLOAT,
                /*int                    */ n,
                /*int                    */ co,
                /*int                    */ ho,
                /*int                    */ wo));

    hipdnnConvolutionBwdFilterAlgo_t algo;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(
                /*hipdnnHandle_t                        */ handle,
                /*const hipdnnTensorDescriptor_t        */ xDesc,
                /*const hipdnnTensorDescriptor_t        */ dyDesc,
                /*const hipdnnConvolutionDescriptor_t   */ convDesc,
                /*const hipdnnFilterDescriptor_t        */ dwDesc,
                /*hipdnnConvolutionBwdFilterPreference_t*/ HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
                /*size_t (is not used)                 */ 0,
                /*hipdnnConvolutionBwdFilterAlgo_t      */ &algo));
    std::cout << "hipdnnConvolutionBwdFilterAlgo_t: " << getAlgoName(algo) << std::endl;

    size_t workSpaceSizeInBytes;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
                /*hipdnnHandle_t                     */ handle,
                /*const hipdnnTensorDescriptor_t     */ xDesc,
                /*const hipdnnTensorDescriptor_t     */ dyDesc,
                /*const hipdnnConvolutionDescriptor_t*/ convDesc,
                /*const hipdnnFilterDescriptor_t     */ dwDesc,
                /*hipdnnConvolutionBwdFilterAlgo_t   */ algo,
                /*size_t                            */ &workSpaceSizeInBytes));

    std::vector<float> h_x(n * ci * hi * wi, 0);
    std::vector<float> h_dy(n * co * ho * wo, 0);
    std::vector<float> h_dw(co * ci * kernel_h * kernel_w,
            std::numeric_limits<float>::quiet_NaN());
    std::vector<float> h_dw_expct(co * ci * kernel_h * kernel_w,
            std::numeric_limits<float>::quiet_NaN());
    rand_vector_float(h_x);
    rand_vector_float(h_dy);

    void *x = nullptr, *dy = nullptr, *dw = nullptr, *workSpace = nullptr;
    size_t size_x = n * ci * hi * wi * sizeof(float);
    size_t size_dy = n * co * ho * wo * sizeof(float);
    size_t size_dw = co * ci * kernel_h * kernel_w * sizeof(float);
    hipMalloc (&x, size_x);
    hipMalloc (&dy, size_dy);
    hipMalloc (&dw, size_dw);
    hipMalloc (&workSpace, workSpaceSizeInBytes);
    hipMemset (dw, 0xff, size_dw);

    hipMemcpy(x, h_x.data(), size_x, hipMemcpyHostToDevice);
    hipMemcpy(dy, h_dy.data(), size_dy, hipMemcpyHostToDevice);

    const float alpha = 1, beta = 0;
    hipEventRecord(start);
    checkCUDNN(hipdnnConvolutionBackwardFilter(
                /*hipdnnHandle_t                     */ handle,
                /*const void *                      */ &alpha,
                /*const hipdnnTensorDescriptor_t     */ xDesc,
                /*const void *                      */ x,
                /*const hipdnnTensorDescriptor_t     */ dyDesc,
                /*const void *                      */ dy,
                /*const hipdnnConvolutionDescriptor_t*/ convDesc,
                /*hipdnnConvolutionBwdFilterAlgo_t   */ algo,
                /*void *                            */ workSpace,
                /*size_t                            */ workSpaceSizeInBytes,
                /*const void *                      */ &beta,
                /*const hipdnnFilterDescriptor_t     */ dwDesc,
                /*void *                            */ dw));
    hipEventRecord(stop);
    pseudoConvolutionBackwardFilter(h_x, h_dy, h_dw_expct,
            n, ci, hi, wi,
            co, ho, wo,
            kernel_h, kernel_w, u, v,
            pad_h, pad_w);

    CHECK(hipDeviceSynchronize());
    hipMemcpy(h_dw.data(), dw, size_dw, hipMemcpyDeviceToHost);

    std::cout << "Max Abs Error(expect vs actual): "
              << getMaxAbsError(h_dw_expct, h_dw) << std::endl;

    float msec = 0;
    hipEventElapsedTime(&msec, start, stop);
    std::cout << "Exec time: " << msec * 1000 << "[usec]" << std::endl;

    hipFree(x);
    hipFree(dy);
    hipFree(dw);
    hipFree(workSpace);
    hipdnnDestroyTensorDescriptor(xDesc);
    hipdnnDestroyTensorDescriptor(dyDesc);
    hipdnnDestroyFilterDescriptor(dwDesc);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipdnnDestroy(handle);
    CHECK(hipDeviceSynchronize());
    return 0;
}

std::mt19937 mt(0);
void rand_vector_float (std::vector<float> &v) {
    std::normal_distribution<> rand(0, 5);
    for (std::vector<float>::iterator i = v.begin(); i != v.end(); ++i) {
        *i = rand(mt);
    }
    return;
}

namespace {
    const char* getAlgoName(hipdnnConvolutionBwdFilterAlgo_t algo) 
    {
        switch (algo) 
        {
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0"; /* non-deterministic */
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1";
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT";
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_3: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_3"; /* non-deterministic */
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_WINOGRAD: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_WINOGRAD"; /* not implemented */
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_WINOGRAD_NONFUSED: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_WINOGRAD_NONFUSED";
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT_TILING: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT_TILING";
            case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT: return "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT";
            default: std::exit(EXIT_FAILURE); return "Error";
        }
    }
}

namespace {
    int getIndex(
            const int n,
            const int c,
            const int h,
            const int w,
            const int N,
            const int C,
            const int H,
            const int W
            ) {
        return n * C * H * W
            + c * H * W
            + h * W
            + w;
    }
}

void pseudoConvolutionBackwardFilter(
        const std::vector<float> &x, 
        const std::vector<float> &dy,
        std::vector<float> &dw,
        const int N, const int  Ci, const int Hi, const int Wi,
        const int Co, const int Ho, const int Wo,
        const int Hk, const int Wk, const int Hs, const int Ws,
        const int Hp, const int Wp
        ) {

    for (std::vector<float>::iterator i = dw.begin(); i != dw.end(); ++i)
        *i = 0.f;

    int idx_x, idx_dy, idx_dw;
    for (int hi = 0; hi < Hi; ++hi) {
        for (int ho = 0; ho < Ho; ++ho) {
            for (int hk = 0; hk < Hk; ++hk) {
                if ((ho * Hs + hk) != (hi + Hp)) {
                    continue;
                }
                for (int wi = 0; wi < Wi; ++wi) {
                    for (int wo = 0; wo < Wo; ++wo) {
                        for (int wk = 0; wk < Wk; ++wk) {
                            if ( (wo * Ws + wk) != (wi + Wp)) {
                                continue;
                            }
                            for (int n = 0; n < N; ++n) {
                                for (int ci = 0; ci < Ci; ++ci) {
                                    for (int co = 0; co < Co; ++co) {
                                        idx_x  = getIndex(n, ci, hi, wi, N, Ci, Hi, Wi);
                                        idx_dy = getIndex(n, co, ho, wo, N, Co, Ho, Wo);
                                        idx_dw = getIndex(co, ci, hk, wk, Co, Ci, Hk, Wk);
                                        dw[idx_dw] += x[idx_x] * dy[idx_dy];
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    return;
}

float getMaxAbsError(const std::vector<float> &exp, const std::vector<float> &act) {
    float tmp, ret;
    std::vector<float> abs_err(exp.size(),0.f);
    for (std::vector<float>::iterator i = abs_err.begin(); i != abs_err.end(); ++i) {
        size_t index = std::distance(abs_err.begin(), i);
        tmp  = act[index] - exp[index];
        *i = (tmp >= 0) ? tmp : -tmp;
    }

    ret = *std::max_element(abs_err.begin(), abs_err.end());
    return ret;
}
